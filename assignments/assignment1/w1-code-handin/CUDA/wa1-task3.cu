#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void squareKernel(float* d_in, float *d_out) {
    const unsigned int lid = threadIdx.x; // local id inside a block
    const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
    d_out[gid] = pow((d_in[gid]/(d_in[gid] - 2.3)), 3.0); // do computation
}

int main(int argc, char** argv) {
    unsigned int N = atoi(argv[1]);
    unsigned int mem_size = N*sizeof(float);

    // allocate host memory
    float* h_in = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    // initialize the memory
    for (unsigned int i = 0; i < N; ++i) {
        h_in[i] = (float)i;
    }

    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in, mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // execute the kernel
    squareKernel<<< (256 - 1 + N)/256, 256>>>(d_in, d_out);

    // copy result from device to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print result
    for (unsigned int i = 0; i < N; ++i) {
        printf("%d - %.6f\n", i, h_out[i]);
    }

    // clean-up memory
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
}