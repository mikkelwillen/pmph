#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void squareKernel(float* d_in, float *d_out) {
    const unsigned long lid = threadIdx.x; // local id inside a block
    const unsigned long gid = blockIdx.x*blockDim.x + lid; // global id
    d_out[gid] = pow((d_in[gid]/(d_in[gid] - 2.3)), 3.0); // do computation
}

int main(int argc, char** argv) {
    unsigned long N = 753411;
    unsigned long mem_size = N*sizeof(float);

    // allocate host memory
    float* h_in = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    // initialize the memory
    for (unsigned long i = 0; i < N; ++i) {
        h_in[i] = (float)i;
    }

    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in, mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // execute the kernel
    squareKernel<<< 1, N>>>(d_in, d_out);

    // copy result from device to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print result
    for (unsigned long i = 0; i < N; ++i) {
        printf("%d - %.6f\n", i, h_out[i]);
    }

    // clean-up memory
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
}